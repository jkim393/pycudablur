
#include <hip/hip_runtime.h>
__global__ void gaussian_blur(const unsigned char *inputChannel, unsigned char *outputChannel,
                             const unsigned int width, const unsigned int height,
                             const float *gaussianKernel, const unsigned int filterWidth) {
    const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    if(row < height && col < width) {
        const int filterHalf = filterWidth / 2;
        float blur = 0.0;
        for(int i = -filterHalf; i <= filterHalf; i++) {
            for(int j = -filterHalf; j <= filterHalf; j++) {
                const unsigned int y = max(0, min(height - 1, row + i));
                const unsigned int x = max(0, min(width - 1, col + j));

                const float w = gaussianKernel[(j + filterHalf) + (i + filterHalf) * filterWidth];
                blur += w * inputChannel[x + y * width];
            }
        }
        outputChannel[col + row * width] = static_cast<unsigned char>(blur);
    }
}
